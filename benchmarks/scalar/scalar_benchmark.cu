#include "hip/hip_runtime.h"

#include <benchmark/benchmark.h>
#include <iostream>
#include <hipcub/hipcub.hpp> 
#include <thrust/iterator/iterator_traits.h>

template <std::size_t block_size, typename T>
__global__ void simple_reduction(std::size_t input_size, T *global_result) {

  auto tid = threadIdx.x + blockIdx.x * gridDim.x;

  T thread_data{};

  while (tid < input_size) {
    ++thread_data;
    tid += blockDim.x * gridDim.x;
  }

  using BlockReduce = hipcub::BlockReduce<T, block_size>;

  __shared__ typename BlockReduce::TempStorage temp_storage;

  T block_result = BlockReduce(temp_storage).Sum(thread_data);

  if (threadIdx.x == 0)
    atomicAdd(global_result, block_result);
}

template <typename T> static void BM_device_memory(::benchmark::State &state) {

  auto size = 1'000'000;

  T *d_result{};

  hipMalloc(&d_result, sizeof(T));

  constexpr std::size_t block_size{256};

  auto grid_size = (size + block_size + 1) / size;

  for (auto _ : state) {
    hipMemset(d_result, 0, sizeof(T));
    simple_reduction<block_size><<<block_size, grid_size>>>(size, d_result);
    hipDeviceSynchronize();
  }
}
BENCHMARK_TEMPLATE(BM_device_memory, int)
    ->RangeMultiplier(2)
    ->Range(1 << 10, 1 << 18);

#include "hip/hip_runtime.h"


#include "kernels.cuh"

#include <benchmark/benchmark.h>
#include <cuda/atomic>
#include <new>

template <std::size_t block_size>
__global__ void std_reduction(
    std::size_t input_size,
    int *global_result,
    int *device_result,
    int *count)
{
   auto tid = threadIdx.x + blockIdx.x * gridDim.x;
   int thread_data{};
   while (tid < input_size)
   {
      ++thread_data;
      tid += blockDim.x * gridDim.x;
   }
   using BlockReduce = hipcub::BlockReduce<int, block_size>;
   __shared__ typename BlockReduce::TempStorage temp_storage;
   int block_result = BlockReduce(temp_storage).Sum(thread_data);
   bool is_last_block_done = false;

   cuda::atomic_ref<int, cuda::thread_scope_device> device_result_ref{*device_result};
   cuda::atomic_ref<int, cuda::thread_scope_device> count_ref{*count};
   if (threadIdx.x == 0)
   {
      device_result_ref.fetch_add(block_result, cuda::std::memory_order_relaxed);
      unsigned value = count_ref.fetch_add(1, cuda::memory_order_release);
      is_last_block_done = value == (gridDim.x - 1);
      if (is_last_block_done)
      {
         // copy result to global buffer
         cuda::atomic_ref<int, cuda::thread_scope_system> global_result_ref{*global_result};
         global_result_ref.store(device_result_ref.load(cuda::std::memory_order_relaxed), cuda::std::memory_order_relaxed);
         
         
         device_result_ref.store(0, cuda::std::memory_order_relaxed); // set to zero for next timek
         count_ref.store(0, cuda::std::memory_order_relaxed);  // set to zero for next time
      }
   }
}

template <typename T>
static void BM_std_pinned_memory(::benchmark::State &state)
{
   auto size = state.range(0);

   int *storage;
   hipMalloc(&storage, 2 * sizeof(int));

   int *hd_result{};
   hipHostMalloc(&hd_result, sizeof(int));
   T h_result{};
   constexpr std::size_t block_size{256};
   auto grid_size = (size + block_size + 1) / size;

   for (auto _ : state)
   {
      *hd_result = 0;
      std_reduction<block_size><<<block_size, grid_size>>>(size, hd_result, &storage[0], &storage[1]);
      cuda::atomic_ref<int, cuda::thread_scope_system> hd_result_ref{*hd_result};
      while (hd_result_ref.load(cuda::memory_order_acquire) == 0);
      benchmark::DoNotOptimize(h_result = *hd_result);
   }

   hipHostFree(hd_result);
   hipFree(storage);
}
BENCHMARK_TEMPLATE(BM_std_pinned_memory, int)
    ->Apply(generate_size)
    ->Unit(benchmark::kMicrosecond);
